#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#include <stdio.h>
#include <assert.h>

static const int NTHREADS = 32;

template <typename Dtype>
__global__ void cunn_WeightedClassNLLCriterion_updateOutput_kernel1(Dtype *output,
                                                           Dtype *total_weight,
                                                           Dtype *input,
                                                           THCIndex_t  *target,
                                                           Dtype *weight_map,
                                                           Dtype *weights,
                                                           int size_average,
                                                           int n_classes) {
  assert(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel.

  int t = (int)*target - TH_INDEX_BASE;
  assert(t >= 0 && t < n_classes);
  Dtype cur_weight = weights ? weights[t] : ScalarConvert<int, Dtype>::to(1);
  cur_weight = cur_weight * weight_map[0];
  *output = -cur_weight * input[t];
  *total_weight = cur_weight;
  if (size_average && *total_weight > 0) {
    *output /= *total_weight;
  }
}

template <typename Dtype, typename Acctype>
__global__ void cunn_WeightedClassNLLCriterion_updateOutput_kernel(Dtype *output,
                                                           Dtype *total_weight,
                                                           Dtype *input,
                                                           THCIndex_t *target,
                                                           Dtype *weight_map,
                                                           Dtype *weights,
                                                           int size_average,
                                                           int nframe,
                                                           int ndim,
                                                           int n_classes) {
  __shared__ Acctype shInputs[NTHREADS], acc_weight[NTHREADS];
  int i, t;
  Dtype cur_weight;

  shInputs[threadIdx.x] = ScalarConvert<int, Acctype>::to(0);
  acc_weight[threadIdx.x] = ScalarConvert<int, Acctype>::to(0);
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
      t = target[i] - TH_INDEX_BASE;
      assert(t >= 0 && t < n_classes);
      cur_weight = weights ? weights[t] : ScalarConvert<int, Dtype>::to(1);
      cur_weight = cur_weight * weight_map[i];
      shInputs[threadIdx.x] -= input[i * ndim + t] * cur_weight;
      acc_weight[threadIdx.x] += cur_weight;
  }
  __syncthreads();

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel

  if (threadIdx.x == 0) {
    *output = *total_weight = ScalarConvert<int, Dtype>::to(0);
    Acctype outputAcc = 0;
    Acctype total_weightAcc = 0;
    for (i = 0; i < NTHREADS; ++i){
      // FIXME should we do somethigng here
      outputAcc += shInputs[i];
      total_weightAcc += acc_weight[i];
    }
    *total_weight = ScalarConvert<Acctype, Dtype>::to(total_weightAcc);
    *output = ScalarConvert<Acctype, Dtype>::to(outputAcc);
    if (size_average && *total_weight > 0) {
      *output = ScalarConvert<Acctype, Dtype>::to(outputAcc / total_weightAcc);
    }

  }
}

template <typename Dtype>
__global__ void cunn_WeightedClassNLLCriterion_updateGradInput_kernel1(
  Dtype* gradInput,
  Dtype* weights,
  THCIndex_t* target,
  Dtype* weight_map,
  Dtype* total_weight,
  int size_average,
  int n_classes)
{
  Dtype cur_weight;
  if (*total_weight <= 0) {
    return;
  }
  Dtype norm = size_average ? (ScalarConvert<int, Dtype>::to(1) / *total_weight) : ScalarConvert<int, Dtype>::to(1);
  int t = (int)*target - TH_INDEX_BASE;
  assert(t >= 0 && t < n_classes);
  cur_weight = weights ? weights[t] : ScalarConvert<int, Dtype>::to(1);
  cur_weight = cur_weight * weight_map[0];
  gradInput[t] = -(cur_weight) * norm;
}

template <typename Dtype>
__global__ void cunn_WeightedClassNLLCriterion_updateGradInput_kernel(
  Dtype *gradInput,
  THCIndex_t *target,
  Dtype *weight_map,
  Dtype *weights,
  Dtype *total_weight,
  int size_average,
  int nframe,
  int ndim,
  int n_classes)
{
  if (*total_weight <= 0) {
    return;
  }
  int i, t;
  Dtype cur_weight;
  Dtype norm = size_average ? (ScalarConvert<int, Dtype>::to(1) / *total_weight) : ScalarConvert<int, Dtype>::to(1);

  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    t = (int)target[i] - TH_INDEX_BASE;
    assert(t >= 0 && t < n_classes);
    cur_weight = weights ? weights[t] : ScalarConvert<int, Dtype>::to(1);
    cur_weight = cur_weight * weight_map[i];
    gradInput[i * ndim + t] = -(cur_weight) * norm;
  }
}

#include "generic/WeightedClassNLLCriterion.cu"
#include "THCGenerateFloatTypes.h"
